#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include "IO.h"
#include <cutil_inline.h>

//Device Code
__global__ void Binariza(float *Tratada, float *Original, unsigned int nDimX, unsigned int nDimY, float Media)
{
	unsigned int i;

	// Creamos el indice del hilo
	i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if( i <= (nDimX * nDimY-1) ){
		if (Original[i] <= Media)
			Tratada[i]=255;
		else
			Tratada[i]=0;
	}
}





int main(int argc, char* argv[]) {
   unsigned char *Cabecera=NULL;
   char          *Entrada=NULL;
   char          *Salida=NULL;

   unsigned int   nDimX, nDimY, CabeceraSize, rc, Itera, i;
   unsigned short BytePorPixel;

   int HilosPorBloque=0, BloquesPorGrid=0, media=0;

   float *ImagenTratada=NULL,    *ImagenOriginal=NULL;
   float *GPUTratada=NULL, *GPUOriginal=NULL;
   float time=.0; 

   hipEvent_t start, stop;

   if (argc != 6) {
      printf("Uso: Suaviza-Secuencial <Fichero-Entrada> <Fichero-Salida> <Iteraciones> <Threads per Block> <Media>\n");
      return -1;
   }

   Entrada        = strdup(argv[1]);
   Salida         = strdup(argv[2]);
   Itera          = atoi(argv[3]);
   HilosPorBloque = atoi(argv[4]);
   media 		  = atoi(argv[5]);
   
   /* Obtener caracteristicas de la imagen */
   rc = nGetSize(Entrada, &nDimX, &nDimY, &BytePorPixel, &CabeceraSize);
   assert(!rc);

   /* Asegurando que es 1 Byte por Pixel */
   if (BytePorPixel != 1) {
      printf("Lo sentimos pero solo para 1 Byte por pixel\n");
      return -1;
   }

   /* Para la Cabecera de la Imagen */
   Cabecera = (unsigned char *)calloc(CabeceraSize, sizeof(unsigned char));
   assert(Cabecera);

   /* Almacena la Imagen que se esta actualizado (t+1) */
   ImagenTratada = (float *)calloc(BytePorPixel*nDimX*nDimY, sizeof(float));
   assert(ImagenTratada);

   /* Almacena la Imagen que se usa para actualizar en (t+1), esto es, la imagen en (t) */
   ImagenOriginal  = (float *)calloc(BytePorPixel*nDimX*nDimY, sizeof(float));
   assert(ImagenOriginal);

   /* Leer la imagen */
   rc = nLoad3(Entrada, nDimX, nDimY, ImagenOriginal, Cabecera, CabeceraSize);
   assert(!rc);

   /* No usar llamadas a cuda antes de usar setdevice. */
   //cutilSafeCall(hipSetDevice(0));

   hipEventCreate(&start);
   hipEventCreate(&stop);

   // Reservando memoria para los vectores en el device
   // cutilSafeCall( );
   hipMalloc((void**)&GPUOriginal,  nDimX*nDimY*sizeof(float));
   hipMalloc((void**)&GPUTratada, nDimX*nDimY*sizeof(float));

   //cutilSafeCall(
   hipMemcpy(GPUOriginal, ImagenOriginal, nDimX*nDimY*sizeof(float), hipMemcpyHostToDevice);

   BloquesPorGrid= (nDimX*nDimY + HilosPorBloque -1) / HilosPorBloque;

   hipEventRecord(start,0);
	printf("comienza");
   for (i=1; i<=Itera; i++)
   {
      if ((i % 2) != 0)
         Binariza<<<BloquesPorGrid, HilosPorBloque>>>(GPUTratada,  GPUOriginal,  nDimX, nDimY, media);
      else
         Binariza<<<BloquesPorGrid, HilosPorBloque>>>(GPUOriginal, GPUTratada, nDimX, nDimY, media);
   }
   printf("acaba");
   //cutilCheckMsg("kernel launch failure");
   
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time,start,stop);

   printf("tiempo %f\n", time);

   if ((Itera % 2) != 0)
      // cutilSafeCall();
 	hipMemcpy(ImagenTratada, GPUTratada, nDimX*nDimY*sizeof(float), hipMemcpyDeviceToHost);
   else
      // cutilSafeCall();
	hipMemcpy(ImagenTratada, GPUOriginal,  nDimX*nDimY*sizeof(float), hipMemcpyDeviceToHost);

   /* Guardar la imagen */
   rc = nSave3(Salida, nDimX, nDimY, ImagenTratada, Cabecera, CabeceraSize);
   assert(!rc);
   
   free(Cabecera);
   free(ImagenTratada);
   free(ImagenOriginal);

//   cutilSafeCall();
	hipFree(GPUTratada);
   	hipFree(GPUOriginal);
   hipEventDestroy(start);
   hipEventDestroy(stop);  

   return 0;
}
