#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include "IO.h"
#include "Filtros.h"
#include "cutil.h"

__global__ void Suma(float* A, float* B, float* C)
{
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char* argv[]) {
   unsigned char *Cabecera=NULL;
   char          *Entrada=NULL;
   char          *Salida=NULL;

   unsigned int   nDimX, nDimY, CabeceraSize, rc, Itera, i;
   unsigned short BytePorPixel;
   float         **Imagen, *tImagen=NULL , *aImagen=NULL;
   float         **d_A, **d_B;

   if (argc != 4) {
      printf("Uso: Suaviza-Secuencial <Fichero-Entrada> <Fichero-Salida> Iteraciones\n");
      return -1;
   }

   Entrada = strdup(argv[1]);
   Salida  = strdup(argv[2]);
   Itera   = atoi(argv[3]);
   
   /* Obtener caracteristicas de la imagen */
   rc = nGetSize(Entrada, &nDimX, &nDimY, &BytePorPixel, &CabeceraSize);
   assert(!rc);

   /* Asegurando que es 1 Byte por Pixel */
   if (BytePorPixel != 1) {
      printf("Lo sentimos pero solo para 1 Byte por pixel\n");
      return -1;
   }

   /* Para la Cabecera de la Imagen */
   Cabecera = (unsigned char *)calloc(CabeceraSize, sizeof(unsigned char));
   assert(Cabecera);

   /* Para el Cuerpo de la Imagen */
   Imagen = Crea_Matriz(nDimY,nDimX);
   assert(Imagen);

   /* Vectores para almacenamiento temporal */
   tImagen = (float *)calloc(BytePorPixel*nDimX, sizeof(float));
   assert(tImagen);
   aImagen = (float *)calloc(BytePorPixel*nDimX, sizeof(float));
   assert(aImagen);

   /* Leer la imagen */
   rc = nLoad(Entrada, nDimX, nDimY, Imagen, Cabecera, CabeceraSize);
   assert(!rc);

   /* Reservar memoria y ponerla a cero en el device */
   unsigned int size_A = nDimX * nDimY;
   unsigned int mem_size_A = sizeof(float) * size_A;

   unsigned int size_B = nDimX * nDimY;
   unsigned int mem_size_B = sizeof(float) * size_B;
   
   CUDACreaMatriz(nDimY, nDimX, d_A);
   CUDACreaMatriz(nDimY, nDimX, d_B);

   cutilSafeCall(hipMemcpy(d_A, Imagen, mem_size_A, hipMemcpyHostToDevice));
   cutilSafeCall(hipMemcpy(d_B, Imagen, mem_size_A, hipMemcpyHostToDevice));




   
   /* Guardar la imagen */
   rc = nSave(Salida, nDimX, nDimY, Imagen, Cabecera, CabeceraSize);
   assert(!rc);
   
   free(Cabecera);
   Liberar_Matriz(Imagen, nDimY);
   free(tImagen);
   free(aImagen);

   return 0;
}
