/*---------------------------------------------------------------*/
/* example 02  - Device Management                               */
/* Description : get properties of all visible device/GPU        */
/* Version     : 1.0 for CUDA 2.0                                */
/* Compilation :                                                 */
/*   nvcc --host-compilation C -o example02 example02.cu         */
/* Execution   :                                                 */
/*   example02                                                   */
/*---------------------------------------------------------------*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

int main (int argc, char *argv[] ) {
  int nbDevice, dev;
  struct hipDeviceProp_t prop;

  hipGetDeviceCount( &nbDevice );
  printf( "Number of device: %d\n", nbDevice );

  for( dev = 0 ; dev < nbDevice ; ++dev ) {
    printf( "\nProperties of CUDA device number %d\n", dev );
    hipGetDeviceProperties( &prop, dev );

    printf( "  Device Name:        %s\n",     prop.name      );
    printf( "  Clock frequency:    %d kHz\n", prop.clockRate );
    printf( "  Compute capability: %d.%d\n",  prop.major, prop.minor );
    printf( "  Device Overlap:     " );
    switch( prop.deviceOverlap ) {
      case 0:
       printf( "No \n" );
      break;
      case 1:
       printf( "Yes\n" );
      break;
      default :
        printf( "Unknown (%d)\n", prop.deviceOverlap );
      break;
    }
    printf( "  Number of Multi-processor: %d\n", 
            prop.multiProcessorCount );
    printf( "  Global Memory Size:        %4d MB\n", 
            (prop.totalGlobalMem / 1048576) );
    printf( "  Memory Maximum Pitch:      %4d kB\n", 
            (prop.memPitch / 1024) );
    printf( "  Texture Alignment:         %4d bytes\n", 
            prop.textureAlignment );
    printf( "  Constant Memory Size:      %4d MB\n", 
            (prop.totalConstMem / 1024) );
    printf( "  Shared Memory Size :       %4d kB per multi-processor\n", 
            (prop.sharedMemPerBlock / 1024) );
    printf( "  Number of 32-bit registers: %d per multi-processor\n", 
            prop.regsPerBlock );
    printf( "  Grid maximum size (X,Y,Z):         %5d x %5d x %5d\n", 
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]  );
    printf( "  Thread bloc maximum size (X,Y,Z):  %5d x %5d x %5d\n", 
            prop.maxThreadsDim[0], 
            prop.maxThreadsDim[1],
            prop.maxThreadsDim[2]  );
    printf( "  Maximum Number of threads per block: %d per block of thread\n", 
            prop.maxThreadsPerBlock );
    printf( "  Warp Size: %d threads\n", prop.warpSize );
  }

  return 0;
}

