#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include "IO.h"
#include <cutil_inline.h>

//Device Code
__global__ void SuavizaCuda(float *Tratada, float *Original, unsigned int nDimX, unsigned int nDimY)
{
  unsigned int
      UpIzq, UpCen, UpDer,
      Izqda, Centr, Derch,
      DoIzq, DoCen, DoDer,
      UltPriFil, PriUltFil, UltUltFil,
      PriCol, UltCol;
   
   float
      fUpIzq, fUpCen, fUpDer,
      fIzqda, fCentr, fDerch,
      fDoIzq, fDoCen, fDoDer,
      Divisor;
 


	// Creamos el indice del hilo
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   UltPriFil=nDimX-1;
   PriUltFil=(nDimY-1)*nDimX;
   UltUltFil=nDimY*nDimX-1;
   
   if( i <= (nDimX * nDimY-1) ){
  
      UpCen=i-nDimX; UpIzq=UpCen-1; UpDer=UpCen+1;
      Centr=i;       Izqda=Centr-1; Derch=Centr+1;
      DoCen=i+nDimX; DoIzq=DoCen-1; DoDer=DoCen+1;

      PriCol=i     % nDimX;
      UltCol=(i+1) % nDimX;
   
      if (i == 0) /* elemento (0,0) */
      {
         fUpIzq=.0; fUpCen=.0;            fUpDer=.0;
         fIzqda=.0; fCentr=Original[Centr]; fDerch=Original[Derch];
         fDoIzq=.0; fDoCen=Original[DoCen]; fDoDer=Original[DoDer];
         Divisor=4.0;
      }
      else
      {
         if (i == UltPriFil) /* elemento (0, nDimX-1) */
         {
            fUpIzq=.0;            fUpCen=.0;            fUpDer=.0;
            fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=.0;
            fDoIzq=Original[DoIzq]; fDoCen=Original[DoCen]; fDoDer=.0;
            Divisor=4.0;
         }
         else
         {
            if (i == PriUltFil) /* elemento (nDimY-1, 0) */
            {
               fUpIzq=.0; fUpCen=Original[UpCen]; fUpDer=Original[UpDer];
               fIzqda=.0; fCentr=Original[Centr]; fDerch=Original[Derch];
               fDoIzq=.0; fDoCen=.0;            fDoDer=.0;
               Divisor=4.0;
            }
            else
            {
               if (i == UltUltFil) /* elemento (nDimY-1, nDimX-1) */ 
               {
                  fUpIzq=Original[UpIzq]; fUpCen=Original[UpCen]; fUpDer=.0;
                  fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=.0;
                  fDoIzq=.0;            fDoCen=.0;            fDoDer=.0;
                  Divisor=4.0;
               }
               else 
               {
                  if (i < UltPriFil) /* elementos intermedios primera fila */
                  {
                     fUpIzq=.0;            fUpCen=.0;            fUpDer=.0;
                     fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=Original[Derch];
                     fDoIzq=Original[DoIzq]; fDoCen=Original[DoCen]; fDoDer=Original[DoDer];
                     Divisor=6.0;
                  }
                  else 
                  {
                     if (i > PriUltFil) /* elementos intermedios ultima fila */
                     {
                        fUpIzq=Original[UpIzq]; fUpCen=Original[UpCen]; fUpDer=Original[UpDer];
                        fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=Original[Derch];
                        fDoIzq=.0;            fDoCen=.0;            fDoDer=.0;
                        Divisor=6.0;
                     }
                     else 
                     {
                        if (PriCol == 0) /* elementos primera columna no tratados ya */
                        {
                           fUpIzq=.0; fUpCen=Original[UpCen]; fUpDer=Original[UpDer];
                           fIzqda=.0; fCentr=Original[Centr]; fDerch=Original[Derch];
                           fDoIzq=.0; fDoCen=Original[DoCen]; fDoDer=Original[DoDer];
                           Divisor=6.0;
                        }
                        else
                        {
                           if (UltCol == 0) /* elementos ultima columnas no tratados ya */
                           {
                              fUpIzq=Original[UpIzq]; fUpCen=Original[UpCen]; fUpDer=.0;
                              fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=.0;
                              fDoIzq=Original[DoIzq]; fDoCen=Original[DoCen]; fDoDer=.0;
                              Divisor=6.0;
                           }
                           else /* caso general */
                           {
                              fUpIzq=Original[UpIzq]; fUpCen=Original[UpCen]; fUpDer=Original[UpDer];
                              fIzqda=Original[Izqda]; fCentr=Original[Centr]; fDerch=Original[Derch];
                              fDoIzq=Original[DoIzq]; fDoCen=Original[DoCen]; fDoDer=Original[DoDer];
                              Divisor=9.0;
			
                           }
                        }
                     }
                  }
               }
            }
         }     
      }
      

		Tratada[i] = (fUpIzq+fUpCen+fUpDer + fIzqda+fCentr+fDerch + fDoIzq+fDoCen+fDoDer) / Divisor;   
	
  } //Fin del if de comprobacion
}





int main(int argc, char* argv[]) {
   unsigned char *Cabecera=NULL;
   char          *Entrada=NULL;
   char          *Salida=NULL;

   unsigned int   nDimX, nDimY, CabeceraSize, rc, Itera, i;
   unsigned short BytePorPixel;

   int HilosPorBloque=0, BloquesPorGrid=0;

   float *ImagenTratada=NULL,    *ImagenOriginal=NULL;
   float *GPUTratada=NULL, *GPUOriginal=NULL;
   float time=.0; 

   hipEvent_t start, stop;

   if (argc != 5) {
      printf("Uso: Suaviza-Secuencial <Fichero-Entrada> <Fichero-Salida> <Iteraciones> <Threads per Block>\n");
      return -1;
   }

   Entrada        = strdup(argv[1]);
   Salida         = strdup(argv[2]);
   Itera          = atoi(argv[3]);
   HilosPorBloque = atoi(argv[4]);
   
   /* Obtener caracteristicas de la imagen */
   rc = nGetSize(Entrada, &nDimX, &nDimY, &BytePorPixel, &CabeceraSize);
   assert(!rc);

   /* Asegurando que es 1 Byte por Pixel */
   if (BytePorPixel != 1) {
      printf("Lo sentimos pero solo para 1 Byte por pixel\n");
      return -1;
   }

   /* Para la Cabecera de la Imagen */
   Cabecera = (unsigned char *)calloc(CabeceraSize, sizeof(unsigned char));
   assert(Cabecera);

   /* Almacena la Imagen que se esta actualizado (t+1) */
   ImagenTratada = (float *)calloc(BytePorPixel*nDimX*nDimY, sizeof(float));
   assert(ImagenTratada);

   /* Almacena la Imagen que se usa para actualizar en (t+1), esto es, la imagen en (t) */
   ImagenOriginal  = (float *)calloc(BytePorPixel*nDimX*nDimY, sizeof(float));
   assert(ImagenOriginal);

   /* Leer la imagen */
   rc = nLoad3(Entrada, nDimX, nDimY, ImagenOriginal, Cabecera, CabeceraSize);
   assert(!rc);

   /* No usar llamadas a cuda antes de usar setdevice. */
   //cutilSafeCall(hipSetDevice(0));

   hipEventCreate(&start);
   hipEventCreate(&stop);

   // Reservando memoria para los vectores en el device
   // cutilSafeCall( );
   hipMalloc((void**)&GPUOriginal,  nDimX*nDimY*sizeof(float));
   hipMalloc((void**)&GPUTratada, nDimX*nDimY*sizeof(float));

   //cutilSafeCall(
   hipMemcpy(GPUOriginal, ImagenOriginal, nDimX*nDimY*sizeof(float), hipMemcpyHostToDevice);

   BloquesPorGrid= (nDimX*nDimY + HilosPorBloque -1) / HilosPorBloque;

   hipEventRecord(start,0);

   for (i=1; i<=Itera; i++)
   {
      if ((i % 2) != 0)
         SuavizaCuda<<<BloquesPorGrid, HilosPorBloque>>>(GPUTratada,  GPUOriginal,  nDimX, nDimY);
      else
         SuavizaCuda<<<BloquesPorGrid, HilosPorBloque>>>(GPUOriginal, GPUTratada, nDimX, nDimY);
   }
   //cutilCheckMsg("kernel launch failure");
   
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time,start,stop);

   printf("tiempo %f\n", time);

   if ((Itera % 2) != 0)
      // cutilSafeCall();
 	hipMemcpy(ImagenTratada, GPUTratada, nDimX*nDimY*sizeof(float), hipMemcpyDeviceToHost);
   else
      // cutilSafeCall();
	hipMemcpy(ImagenTratada, GPUOriginal,  nDimX*nDimY*sizeof(float), hipMemcpyDeviceToHost);

   /* Guardar la imagen */
   rc = nSave3(Salida, nDimX, nDimY, ImagenTratada, Cabecera, CabeceraSize);
   assert(!rc);
   
   free(Cabecera);
   free(ImagenTratada);
   free(ImagenOriginal);

//   cutilSafeCall();
	hipFree(GPUTratada);
   	hipFree(GPUOriginal);
   hipEventDestroy(start);
   hipEventDestroy(stop);  

   return 0;
}
